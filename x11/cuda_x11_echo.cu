#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <memory.h>

// Folgende Definitionen sp�ter durch header ersetzen
typedef unsigned char uint8_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;

// das Hi Word aus einem 64 Bit Typen extrahieren
static __device__ uint32_t HIWORD(const uint64_t &x) {
#if __CUDA_ARCH__ >= 130
	return (uint32_t)__double2hiint(__longlong_as_double(x));
#else
	return (uint32_t)(x >> 32);
#endif
}

// das Lo Word aus einem 64 Bit Typen extrahieren
static __device__ uint32_t LOWORD(const uint64_t &x) {
#if __CUDA_ARCH__ >= 130
	return (uint32_t)__double2loint(__longlong_as_double(x));
#else
	return (uint32_t)(x & 0xFFFFFFFFULL);
#endif
}

#define SPH_C64(x)    ((uint64_t)(x ## ULL))
#define SPH_C32(x)    ((uint32_t)(x ## U))

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

#include "cuda_x11_aes.cu"

__device__ __forceinline__ void AES_2ROUND(
	const uint32_t* __restrict__ sharedMemory,
	uint32_t &x0, uint32_t &x1, uint32_t &x2, uint32_t &x3,
	uint32_t &k0, uint32_t &k1, uint32_t &k2, uint32_t &k3)
{
	uint32_t y0, y1, y2, y3;
	
	aes_round(sharedMemory,
		x0, x1, x2, x3,
		k0,
		y0, y1, y2, y3);

	aes_round(sharedMemory,
		y0, y1, y2, y3,
		x0, x1, x2, x3);

	// hier werden wir ein carry brauchen (oder auch nicht)
	k0++;
}

__device__ __forceinline__ void cuda_echo_round(
	const uint32_t *sharedMemory,
	uint32_t &k0, uint32_t &k1, uint32_t &k2, uint32_t &k3,
	uint32_t *W, int round)
{
	// W hat 16*4 als Abma�e

	// Big Sub Words
#pragma unroll 16
	for(int i=0;i<16;i++)
	{
		int idx = i<<2; // *4
		AES_2ROUND(sharedMemory,
			W[idx+0], W[idx+1], W[idx+2], W[idx+3],
			k0, k1, k2, k3);
	}

	// Shift Rows
#pragma unroll 4
	for(int i=0;i<4;i++)
	{
		uint32_t t;

		/// 1, 5, 9, 13
		t = W[4 + i];
		W[4 + i] = W[20 + i];
		W[20 + i] = W[36 + i];
		W[36 + i] = W[52 + i];
		W[52 + i] = t;

		// 2, 6, 10, 14
		t = W[8 + i];
		W[8 + i] = W[40 + i];
		W[40 + i] = t;
		t = W[24 + i];
		W[24 + i] = W[56 + i];
		W[56 + i] = t;

		// 15, 11, 7, 3
		t = W[60 + i];
		W[60 + i] = W[44 + i];
		W[44 + i] = W[28 + i];
		W[28 + i] = W[12 + i];
		W[12 + i] = t;
	}

	// Mix Columns
#pragma unroll 4
	for(int i=0;i<4;i++) // Schleife �ber je 2*uint32_t
	{
#pragma unroll 4
		for(int j=0;j<4;j++) // Schleife �ber die elemnte
		{
			int idx = j<<2; // j*4

			uint32_t a = W[ ((idx + 0)<<2) + i];
			uint32_t b = W[ ((idx + 1)<<2) + i];
			uint32_t c = W[ ((idx + 2)<<2) + i];
			uint32_t d = W[ ((idx + 3)<<2) + i];

			uint32_t ab = a ^ b;
			uint32_t bc = b ^ c;
			uint32_t cd = c ^ d;

			uint32_t t;
			t = ((ab & 0x80808080) >> 7);
			uint32_t abx = t<<4 ^ t<<3 ^ t<<1 ^ t;
			t = ((bc & 0x80808080) >> 7);
			uint32_t bcx = t<<4 ^ t<<3 ^ t<<1 ^ t;
			t = ((cd & 0x80808080) >> 7);
			uint32_t cdx = t<<4 ^ t<<3 ^ t<<1 ^ t;

			abx ^= ((ab & 0x7F7F7F7F) << 1);
			bcx ^= ((bc & 0x7F7F7F7F) << 1);
			cdx ^= ((cd & 0x7F7F7F7F) << 1);

			W[ ((idx + 0)<<2) + i] = abx ^ bc ^ d;
			W[ ((idx + 1)<<2) + i] = bcx ^ a ^ cd;
			W[ ((idx + 2)<<2) + i] = cdx ^ ab ^ d;
			W[ ((idx + 3)<<2) + i] = abx ^ bcx ^ cdx ^ ab ^ c;
		}
	}
}

__global__ void x11_echo512_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	__shared__ uint32_t sharedMemory[1024];

	aes_gpu_init(sharedMemory);

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint32_t *Hash = (uint32_t*)&g_hash[hashPosition<<3];

		uint32_t W[64];
		uint32_t k0 = 512, k1 = 0, k2 = 0, k3 = 0; // K0 = bitlen
		/* Initialisierung */
#pragma unroll 8
		for(int i=0;i<32;i+=4)
		{
			W[i + 0] = 512;
			W[i + 1] = 0;
			W[i + 2] = 0;
			W[i + 3] = 0;
		}

		// kopiere 32-byte gro�en hash
#pragma unroll 16
		for(int i=0;i<16;i++)
			W[i+32] = Hash[i];
		W[48] = 0x80; // fest
#pragma unroll 10
		for(int i=49;i<59;i++)
			W[i] = 0;
		W[59] = 0x02000000; // fest
		W[60] = k0; // bitlen
		W[61] = k1;
		W[62] = k2;
		W[63] = k3;
		
		for(int i=0;i<10;i++)
		{
			cuda_echo_round(sharedMemory, k0, k1, k2, k3, W, i);
		}

#pragma unroll 8
		for(int i=0;i<32;i+=4)
		{
			W[i  ] ^= W[32 + i    ] ^ 512;
			W[i+1] ^= W[32 + i + 1];
			W[i+2] ^= W[32 + i + 2];
			W[i+3] ^= W[32 + i + 3];
		}

#pragma unroll 16
		for(int i=0;i<16;i++)
			W[i] ^= Hash[i];

		//W[8] ^= 0x10;
		W[27] ^= 0x02000000;
		W[28] ^= k0;

#pragma unroll 16
		for(int i=0;i<16;i++)
			Hash[i] = W[i];
    }
}

// Setup-Funktionen
__host__ void x11_echo512_cpu_init(int thr_id, int threads)
{
	aes_cpu_init();
}

__host__ void x11_echo512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    const int threadsperblock = 256;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    x11_echo512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
    MyStreamSynchronize(NULL, order, thr_id);
}
